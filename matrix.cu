#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 4
 
__global__ void matrixGPU(int *pMatrizA,int *pMatrizB, int *pMatrizResultante, int pColumnasMatrizA, int pFilasMatrizB, int pColumnasMatrizB)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < pColumnasMatrizB && row < pColumnasMatrizA) 
    {
        for(int i = 0; i < pFilasMatrizB; i++) 
        {
            sum += pMatrizA[row * pFilasMatrizB + i] * pMatrizB[i * pColumnasMatrizB + col];
        }
        pMatrizResultante[row * pColumnasMatrizB + col] = sum;
    }
} 

void matrixCPU(int *pMatrizA, int *pMatrizB, int *pMatrizResultante, int pColumnasMatrizA, int pFilasMatrizB, int pColumnasMatrizB) {
    for (int i = 0; i < pColumnasMatrizA; ++i) 
    {
        for (int j = 0; j < pColumnasMatrizB; ++j) 
        {
            int tmp = 0.0;
            for (int h = 0; h < pFilasMatrizB; ++h) 
            {
                tmp += pMatrizA[i * pFilasMatrizB + h] * pMatrizB[h * pColumnasMatrizB + j];
            }
            pMatrizResultante[i * pColumnasMatrizB + j] = tmp;
        }
    }
}

int main(int argc, char const *argv[])
{
    int columnasMatrizA = 4, filasMatrizB = 4, columnasMatrizB = 4; 
    /* Fixed seed for illustration */
    srand(123456987);

    // allocate memory in host RAM, h_cc is used to store CPU result
    int *h_a, *h_b, *h_c, *h_cc;
    hipHostMalloc((void **) &h_a, sizeof(int)*columnasMatrizA*filasMatrizB);
    hipHostMalloc((void **) &h_b, sizeof(int)*filasMatrizB*columnasMatrizB);
    hipHostMalloc((void **) &h_c, sizeof(int)*columnasMatrizA*columnasMatrizB);
    hipHostMalloc((void **) &h_cc, sizeof(int)*columnasMatrizA*columnasMatrizB);

    // Rellenando Matriz A y Matriz B
    for (int i = 0; i < columnasMatrizA; ++i) {
        for (int j = 0; j < filasMatrizB; ++j) {
            h_a[i * n + j] = rand() % 1024;
            h_b[i * k + j] = rand() % 1024;
        }
    }

    float tiempoGPU, tiempoCPU;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Inicializa tiempo para GPU
    hipEventRecord(start, 0);

    int *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(int)*columnasMatrizA*filasMatrizB);
    hipMalloc((void **) &d_b, sizeof(int)*filasMatrizB*columnasMatrizB);
    hipMalloc((void **) &d_c, sizeof(int)*columnasMatrizA*columnasMatrizB);

    hipMemcpy(d_a, h_a, sizeof(int)*columnasMatrizA*filasMatrizB, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int)*filasMatrizB*columnasMatrizB, hipMemcpyHostToDevice);

    unsigned int grid_rows = (columnasMatrizA + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (columnasMatrizB + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    matrixGPU<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, columnasMatrizA, filasMatrizB, columnasMatrizB);    


    hipMemcpy(h_c, d_c, sizeof(int)*columnasMatrizA*columnasMatrizB, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // calcula tiempo de ejecucion del GPU
    hipEventElapsedTime(&tiempoGPU, start, stop);
    printf("GPU time: %f ms.\n\n", tiempoGPU);

    // Inicializa tiempo de ejecucion del CPU
    hipEventRecord(start, 0);

    matrixCPU(h_a, h_b, h_cc, columnasMatrizA, filasMatrizB,columnasMatrizB);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&tiempoCPU, start, stop);
    printf("CPU time: %f ms.\n\n", tiempoCPU);

    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    return 0;
}
