#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 4

__global__ void saxpyGPU(int *pVectorA,int *pVectorB, int *pVectorResultante, int pDimension, int pConstante)
{ 
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n){
        pVectorResultante[i] = pConstante* pVectorA[i] + pVectorB[i];
    }
} 

void saxpyCPU(int *pVectorA, int *pVectorB, int *pVectorResultante, int pDimension,int pConstante) {
    for (int i = 0; i < pDimension; ++i) 
    {
        int tmp = 0.0;
        tmp += pMatrizA[i * pDimension] * pConstante + pMatrizB[i * pDimension];
        pVectorResultante[i * pDimension] = tmp;
    }
}

int main(int argc, char const *argv[])
{
    int columnasMatrizA = 4, filasMatrizB = 4, columnasMatrizB = 4; 
    int dimension = 4, constante = 8;
    /* Fixed seed for illustration */
    srand(123456987);

    // allocate memory in host RAM, h_cc is used to store CPU result
    int *h_a, *h_b, *h_c, *h_cc;
    hipHostMalloc((void *) &h_a, sizeof(int)*dimension);
    hipHostMalloc((void *) &h_b, sizeof(int)*dimension);
    hipHostMalloc((void *) &h_c, sizeof(int)*dimension);
    hipHostMalloc((void *) &h_cc, sizeof(int)*dimension);

    // Rellenando vector A y vector B
    for (int i = 0; i < dimension; ++i) {
            h_a[i * dimension] = rand() % 1024;
            h_b[i * dimension] = rand() % 1024;
    }

    float tiempoGPU, tiempoCPU;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Inicializa tiempo para GPU
    hipEventRecord(start, 0);

    int *d_a, *d_b, *d_c;
    hipMalloc((void *) &d_a, sizeof(int)*dimension);
    hipMalloc((void *) &d_b, sizeof(int)*dimension);
    hipMalloc((void *) &d_c, sizeof(int)*dimension);

    hipMemcpy(d_a, h_a, sizeof(int)*dimension, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int)*dimension, hipMemcpyHostToDevice);

    unsigned int grid_rows = (dimension + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (dimension + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    multiplicacionGPU<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, dimension,constante);    


    hipMemcpy(h_c, d_c, sizeof(int)*dimension, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // calcula tiempo de ejecucion del GPU
    hipEventElapsedTime(&tiempoGPU, start, stop);
    printf("Tiempo de ejecucion GPU: %f ms.\n\n", tiempoGPU);

    // Inicializa tiempo de ejecucion del CPU
    hipEventRecord(start, 0);

    multiplicacionCPU(h_a, h_b, h_cc, dimension, constante);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&tiempoCPU, start, stop);
    printf("Tiempo de ejecucion CPU: %f ms.\n\n", tiempoCPU);

    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    return 0;
}